#include "hip/hip_runtime.h"
/*****************************************************************************/
/* CUDA implementation of Lloyd's quantization algorithm                     */
/* lloyd(points, psize, codebook, csize, stop_criteria =  10e-7)             */
/*		                                                                     */
/*  - IN     - points:        values to be quantized                         */
/*  - IN     - psize:         size of the points array                       */
/*  - IN/OUT - codebook:      initial codebook and final codebook            */
/*  - IN     - csize:         size of the codebook array                     */
/*  - IN     - stop_criteria: typically 10e-7                                */
/*                                                                           */
/*  Doesn't return the partition because it can be easily calculated         */
/*  as the mid-point between codebooks.                                      */
/*****************************************************************************/
void cudaLloyd(float *points, unsigned int psize, float *codebook, unsigned int csize, float stop_criteria) {
	float *d_points;
	unsigned *d_groups;
	float *d_codebook;
	float *partition = (float*)malloc((csize-1)*sizeof(float));;
	unsigned *counts = (unsigned*)calloc(csize,sizeof(unsigned));
	unsigned *d_counts;
	float *sum = (float*)calloc(csize,sizeof(float));
	float *d_sum;
	float dist;
	float reldist;
	float *d_dist;
	float *d_max,max;
	float *d_min,min;

	hipMalloc((void**)&d_points, sizeof(float)*psize);
	hipMalloc((void**)&d_groups, sizeof(unsigned)*psize);
	hipMalloc((void**)&d_codebook, sizeof(float)*csize);
	hipMalloc((void**)&d_counts, sizeof(unsigned)*csize);
	hipMalloc((void**)&d_sum, sizeof(float)*csize);
	hipMalloc((void**)&d_dist, sizeof(float));
	hipMalloc((void**)&d_max, sizeof(float));
	hipMalloc((void**)&d_min, sizeof(float));

	hipMemcpy(d_points, points, sizeof(float) * psize, hipMemcpyHostToDevice);
	hipMemcpy(d_codebook, codebook, sizeof(float)*csize, hipMemcpyHostToDevice);
	hipMemset(d_counts,0,sizeof(unsigned)*csize);
	hipMemset(d_sum,0,sizeof(float)*csize);
	hipMemset(d_dist,0,sizeof(float));
	hipMemcpy(d_max, d_points, sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(d_min, d_points, sizeof(float), hipMemcpyDeviceToDevice);

	// Initial Table
	for(int i = 0; i < csize-1; i++) {
		partition[i] = (codebook[i] + codebook[i+1]) / 2;
	}

	// Assign each point its codebook group
	unsigned int threads = 256;
	unsigned int blocks = (psize - 1) / threads + 1;
	groupKernelMaxMin<<<blocks, threads>>>(d_points, psize, d_codebook, csize, d_groups, d_counts, d_sum, d_dist,d_max, d_min);

	hipMemcpy(&max, d_max, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&min, d_min, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&dist, d_dist, sizeof(float), hipMemcpyDeviceToHost);
	reldist = abs(dist);

	while(reldist > stop_criteria) {
		hipMemcpy(counts, d_counts, sizeof(unsigned)*csize, hipMemcpyDeviceToHost);
		hipMemcpy(sum, d_sum, sizeof(float)*csize, hipMemcpyDeviceToHost);

		// Update codebook
		//can be done in gpu if csize is big enough
		for(int i = 0; i < csize; i++) {
			if(counts[i] != 0) {
				codebook[i] = sum[i] / counts[i];
				counts[i] = 0;
			} else if (i == 0) {
				codebook[i] = (partition[0] + min) / 2;
			} else if (i == csize-1) {
				codebook[i] = (partition[i-1] + max) / 2;
			} else {
				codebook[i] = (partition[i-1] + partition[i]) / 2;
			}
			sum[i] = 0;
		}
		hipMemcpy(d_codebook, codebook, sizeof(float)*csize, hipMemcpyHostToDevice);
		hipMemset(d_counts,0,sizeof(unsigned)*csize);
		hipMemset(d_sum,0,sizeof(float)*csize);

		// Update Table
		for(int i = 0; i < csize-1; i++) {
			partition[i] = (codebook[i] + codebook[i+1]) / 2;
		}

		reldist = dist;
		hipMemset(d_dist,0,sizeof(float));

		// Assign each point its codebook group
		groupKernel<<<blocks, threads>>>(d_points, psize, d_codebook, csize, d_groups, d_counts, d_sum, d_dist);

		//hipMemcpy(groups, d_groups, sizeof(unsigned int) * psize, hipMemcpyDeviceToHost);

		hipMemcpy(&dist, d_dist, sizeof(float), hipMemcpyDeviceToHost);
		dist /= psize;

		reldist = abs(reldist - dist);
	}  // END WHILE

	free(partition);
	free(counts);
	free(sum);
	hipFree(d_points);
	hipFree(d_groups);
	hipFree(d_codebook);
	hipFree(d_counts);
	hipFree(d_sum);
	hipFree(d_dist);
	hipFree(d_max);
	hipFree(d_min);
}
