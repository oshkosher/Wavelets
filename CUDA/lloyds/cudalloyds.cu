
#include <hip/hip_runtime.h>
/*****************************************************************************/
/* CUDA implementation of Lloyd's quantization algorithm                     */
/* lloyd(points, psize, codebook, csize, stop_criteria =  10e-7)             */
/*		                                                                     */
/*  - IN     - points:        values to be quantized                         */
/*  - IN     - psize:         size of the points array                       */
/*  - IN/OUT - codebook:      initial codebook and final codebook            */
/*  - IN     - csize:         size of the codebook array                     */
/*  - IN     - stop_criteria: typically 10e-7                                */
/*  - IN     - points_are_on_gpu: if true, point data is already on the GPU, */
/*                            and 'points' is a device address.              */
/*                                                                           */
/*  Doesn't return the partition because it can be easily calculated         */
/*  as the mid-point between codebooks.                                      */
/*****************************************************************************/
__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__device__ static float atomicMin(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void groupKernelMaxMin(const float *points, unsigned int psize, float *codebook, unsigned int csize, unsigned int *groups, unsigned int *counts, float *sum, float *dist, float *max, float *min) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < psize) {
		float p = points[idx];
		float d = abs(p - codebook[0]);
		float m = d;
		unsigned int g = 0;
		for(int i = 1; i < csize; i++) {
			d = abs(p - codebook[i]);
			bool isminus = (d < m);
			g += isminus * (i - g);
			m -= isminus * (m - d);
		}
		groups[idx] = g;
		atomicAdd(&(counts[g]),1);
		atomicAdd(&(sum[g]),p);
		atomicAdd(dist,codebook[g]-p);
		atomicMax(max,p);
		atomicMin(min,p);
	}
}

__global__ void groupKernel(const float *points, unsigned int psize, float *codebook, unsigned int csize, unsigned int *groups, unsigned int *counts, float *sum, float *dist) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < psize) {
		float p = points[idx];
		float d = abs(p - codebook[0]);
		float min = d;
		unsigned int g = 0;
		for(int i = 1; i < csize; i++) {
			d = abs(p - codebook[i]);
			bool isminus = (d < min);
			g += isminus * (i - g);
			min -= isminus * (min - d);
		}
		groups[idx] = g;
		atomicAdd(&(counts[g]),1);
		atomicAdd(&(sum[g]),p);
		atomicAdd(dist,fabs(codebook[g]-p));
	}
}


void cudaLloyd(const float *points, unsigned int psize, float *codebook, unsigned int csize, float stop_criteria, bool points_are_on_gpu, int *iterations) {
	const float *d_points;
	unsigned *d_groups;
	float *d_codebook;
	float *partition = (float*)malloc((csize-1)*sizeof(float));;
	unsigned *counts = (unsigned*)calloc(csize,sizeof(unsigned));
	unsigned *d_counts;
	float *sum = (float*)calloc(csize,sizeof(float));
	float *d_sum;
	float dist;
	float reldist;
	float *d_dist;
	float *d_max,max;
	float *d_min,min;

        // check if the point data is already on the GPU
        if (points_are_on_gpu) {
          d_points = points;
        } else {
          hipMalloc((void**)&d_points, sizeof(float)*psize);
          hipMemcpy((void*)d_points, points, sizeof(float) * psize, hipMemcpyHostToDevice);
        }

	hipMalloc((void**)&d_groups, sizeof(unsigned)*psize);
	hipMalloc((void**)&d_codebook, sizeof(float)*csize);
	hipMalloc((void**)&d_counts, sizeof(unsigned)*csize);
	hipMalloc((void**)&d_sum, sizeof(float)*csize);
	hipMalloc((void**)&d_dist, sizeof(float));
	hipMalloc((void**)&d_max, sizeof(float));
	hipMalloc((void**)&d_min, sizeof(float));

	hipMemcpy(d_codebook, codebook, sizeof(float)*csize, hipMemcpyHostToDevice);
	hipMemset(d_counts,0,sizeof(unsigned)*csize);
	hipMemset(d_sum,0,sizeof(float)*csize);
	hipMemset(d_dist,0,sizeof(float));
	hipMemcpy(d_max, d_points, sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(d_min, d_points, sizeof(float), hipMemcpyDeviceToDevice);

        if (iterations) *iterations = 0;

	// Initial Table
	for(int i = 0; i < csize-1; i++) {
		partition[i] = (codebook[i] + codebook[i+1]) / 2;
	}

	// Assign each point its codebook group
	unsigned int threads = 256;
	unsigned int blocks = (psize - 1) / threads + 1;
	groupKernelMaxMin<<<blocks, threads>>>(d_points, psize, d_codebook, csize, d_groups, d_counts, d_sum, d_dist,d_max, d_min);

	hipMemcpy(&max, d_max, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&min, d_min, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&dist, d_dist, sizeof(float), hipMemcpyDeviceToHost);
	reldist = abs(dist);

	while(reldist > stop_criteria) {
		if (iterations) ++*iterations;
		hipMemcpy(counts, d_counts, sizeof(unsigned)*csize, hipMemcpyDeviceToHost);
		hipMemcpy(sum, d_sum, sizeof(float)*csize, hipMemcpyDeviceToHost);

		// Update codebook
		//can be done in gpu if csize is big enough
		for(int i = 0; i < csize; i++) {
			if(counts[i] != 0) {
				codebook[i] = sum[i] / counts[i];
				counts[i] = 0;
			} else if (i == 0) {
				codebook[i] = (partition[0] + min) / 2;
			} else if (i == csize-1) {
				codebook[i] = (partition[i-1] + max) / 2;
			} else {
				codebook[i] = (partition[i-1] + partition[i]) / 2;
			}
			sum[i] = 0;
		}
		hipMemcpy(d_codebook, codebook, sizeof(float)*csize, hipMemcpyHostToDevice);
		hipMemset(d_counts,0,sizeof(unsigned)*csize);
		hipMemset(d_sum,0,sizeof(float)*csize);

		// Update Table
		for(int i = 0; i < csize-1; i++) {
			partition[i] = (codebook[i] + codebook[i+1]) / 2;
		}

		reldist = dist;
		hipMemset(d_dist,0,sizeof(float));

		// Assign each point its codebook group
		groupKernel<<<blocks, threads>>>(d_points, psize, d_codebook, csize, d_groups, d_counts, d_sum, d_dist);

		//cudaMemcpy(groups, d_groups, sizeof(unsigned int) * psize, cudaMemcpyDeviceToHost);

		hipMemcpy(&dist, d_dist, sizeof(float), hipMemcpyDeviceToHost);
		dist /= psize;

		reldist = abs(reldist - dist);
	}  // END WHILE

	free(partition);
	free(counts);
	free(sum);
	if (!points_are_on_gpu) hipFree((void*)d_points);
	hipFree(d_groups);
	hipFree(d_codebook);
	hipFree(d_counts);
	hipFree(d_sum);
	hipFree(d_dist);
	hipFree(d_max);
	hipFree(d_min);
}
