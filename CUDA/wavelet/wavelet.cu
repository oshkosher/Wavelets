#include "hip/hip_runtime.h"
/*
 *
 *  Based on Nvidia convolution separable example.
 *
 *
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <cstdio>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "../../cucheck.h"
#include "wavelet.h"

////////////////////////////////////////////////////////////////////////////////
// Convolution kernel storage
////////////////////////////////////////////////////////////////////////////////
__constant__ float c_Kernel[KERNEL_LENGTH*2];


////////////////////////////////////////////////////////////////////////////////
// Row convolution with Low and Hi pass filter
////////////////////////////////////////////////////////////////////////////////
#define ROWS_BLOCKDIM_X 16
#define ROWS_BLOCKDIM_Y 16
#define	ROWS_RESULT_STEPS 1 //8
#define ROWS_HALO_STEPS 1

__global__ void convolutionRowsMirrorHiLoKernel(float *d_Dst, float *d_Src, int imageW, int imageH, int pitch) {
    __shared__ float s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

    //Offset to the left halo edge
    const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;
	const int fidx  = threadIdx.x % 2;

    d_Src += baseY * pitch + baseX;
	const int half = (baseY * pitch + (blockIdx.x * ROWS_BLOCKDIM_X + threadIdx.x))/2;
	d_Dst += half+(fidx*(imageH*pitch)/2) - ROWS_HALO_STEPS * ROWS_BLOCKDIM_X;

    //Load main data
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
    }

    //Load left halo
#pragma unroll

    for (int i = 0; i < ROWS_HALO_STEPS; i++)
    {
		// If HALO is > 1 maybe d_Src[i * ROWS_BLOCKDIM_X - baseX*2]; is not correct for every ROW_HALO_STEP
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : d_Src[i * ROWS_BLOCKDIM_X - baseX*2];
    }

    //Load right halo
#pragma unroll

    for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
    {
		s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : d_Src[i * ROWS_BLOCKDIM_X - (threadIdx.x+1)*2];
    }

    //Compute and store results
    __syncthreads();
#pragma unroll
    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        float sum = 0;

#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
			sum += c_Kernel[fidx * KERNEL_LENGTH + KERNEL_RADIUS + j] * s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];
        }
		d_Dst[i * ROWS_BLOCKDIM_X] = sum;
    }
}

__global__ void invConvolutionRowsMirrorHiLoKernel(float *d_Dst, float *d_Src, int imageW, int imageH, int pitch) {
    __shared__ float s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

    //Offset to the left halo edge
    const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;
	const int fidx  = threadIdx.x % 2;

	const int half = (baseY * pitch + (blockIdx.x * ROWS_BLOCKDIM_X + threadIdx.x))/2;
	d_Src += half+(fidx*(imageH*pitch)/2) - ROWS_HALO_STEPS * ROWS_BLOCKDIM_X;
	d_Dst += baseY * pitch + baseX;

    //Load main data
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
    }

    //Load left halo
#pragma unroll

    for (int i = 0; i < ROWS_HALO_STEPS; i++)
    {
		// If HALO is > 1 maybe d_Src[i * ROWS_BLOCKDIM_X - baseX*2]; is not correct for every ROW_HALO_STEP
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : d_Src[i * ROWS_BLOCKDIM_X - baseX*2];
    }

    //Load right halo
#pragma unroll

    for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
    {
		s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : d_Src[i * ROWS_BLOCKDIM_X - (threadIdx.x+1)*2];
    }

    //Compute and store results
    __syncthreads();
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        float sum = 0;

#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
			sum += c_Kernel[fidx * KERNEL_LENGTH + KERNEL_RADIUS + j] * s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];
        }

		d_Dst[i * ROWS_BLOCKDIM_X] = sum;
    }
}

void fwt_1D(float **data, const unsigned level, const unsigned nx, const unsigned ny) {
    assert(ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= KERNEL_RADIUS);
    assert(nx % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0);
    assert(ny % ROWS_BLOCKDIM_Y == 0);

	const int mem_size = nx*ny*sizeof(float);

	float *data1, *data2, *aux;
	data1 = *data;
	hipMalloc(&data2, mem_size);

	unsigned w = nx;

    dim3 blocks(nx / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X), ny / ROWS_BLOCKDIM_Y);
    dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);
	convolutionRowsMirrorHiLoKernel<<<blocks, threads>>>(data2, data1, w, ny, w);
	CUCHECK(hipGetLastError());


	for (unsigned i = 1; i < level; i++) {
		blocks.x /= 2;
		w /= 2;

		aux = data2;
		data2 = data1;
		data1 = aux;

		hipMemcpy(data2+w*ny, data1+w*ny, w*ny*sizeof(float), hipMemcpyDeviceToDevice);

		convolutionRowsMirrorHiLoKernel<<<blocks, threads>>>(data2, data1, w, ny, w);
                CUCHECK(hipGetLastError());
	}
    
	*data = data2;
	hipFree(data1);

	printf("Rows fwt_1D: %s\n",hipGetErrorString(hipGetLastError()));
}

void iwt_1D(float **data, const unsigned level, const unsigned nx, const unsigned ny) {
    assert(ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= KERNEL_RADIUS);
    assert(nx % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0);
    assert(ny % ROWS_BLOCKDIM_Y == 0);

	const int mem_size = nx*ny*sizeof(float);

	float *data1, *data2, *aux;
	data1 = *data;
	hipMalloc(&data2, mem_size);

	unsigned w = nx >> (level-1);

    dim3 blocks(w / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X), ny / ROWS_BLOCKDIM_Y);
    dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);
	invConvolutionRowsMirrorHiLoKernel<<<blocks, threads>>>(data2, data1, w, ny, w);
        CUCHECK(hipGetLastError());

	for (unsigned i = 1; i < level; i++) {
		hipMemcpy(data2+w*ny, data1+w*ny, (nx-w)*ny*sizeof(float), hipMemcpyDeviceToDevice);
		
		blocks.x *= 2;
		w *= 2;

		aux = data2;
		data2 = data1;
		data1 = aux;

		invConvolutionRowsMirrorHiLoKernel<<<blocks, threads>>>(data2, data1, w, ny, w);
                CUCHECK(hipGetLastError());
	}
    
	*data = data2;
	hipFree(data1);

	printf("Rows iwt_1D: %s\n",hipGetErrorString(hipGetLastError()));
}

////////////////////////////////////////////////////////////////////////////////
// Transpose
////////////////////////////////////////////////////////////////////////////////

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

__global__ void transposeDiagonal(float *odata, const float *idata, int width, int height) {
	__shared__ float tile[TILE_DIM][TILE_DIM+1];

	int blockIdx_x, blockIdx_y;

	// diagonal reordering
	if (width == height) {
		blockIdx_y = blockIdx.x;
		blockIdx_x = (blockIdx.x+blockIdx.y)%gridDim.x;
	} else {
		int bid = blockIdx.x + gridDim.x*blockIdx.y;
		blockIdx_y = bid%gridDim.y;
		blockIdx_x = ((bid/gridDim.y)+blockIdx_y)%gridDim.x;
	}

	int xIndex = blockIdx_x*TILE_DIM + threadIdx.x;
	int yIndex = blockIdx_y*TILE_DIM + threadIdx.y;
	int index_in = xIndex + (yIndex)*width;

	xIndex = blockIdx_y*TILE_DIM + threadIdx.x;
	yIndex = blockIdx_x*TILE_DIM + threadIdx.y;
	int index_out = xIndex + (yIndex)*height;

	for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
		tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
	}

	__syncthreads();

	for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
		odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];
	}
} 
void transpose(float *tdata, const float *idata, const unsigned nx, const unsigned ny) {

	dim3 grid(nx/TILE_DIM, ny/TILE_DIM);
	dim3 threads(TILE_DIM,BLOCK_ROWS); 
	transposeDiagonal<<<grid, threads>>>(tdata, idata, nx, ny);
        CUCHECK(hipDeviceSynchronize());
}

extern "C" void setUpFilter(const float *filter){

    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), filter, KERNEL_LENGTH*2 * sizeof(float));

	printf("Setup: %s\n",hipGetErrorString(hipGetLastError()));
}

extern "C" void fwt_1D_GPU(float *data, const unsigned level, const unsigned nx, const unsigned ny) {
	const int mem_size = nx*ny*sizeof(float);

	float *d_idata;
	hipMalloc(&d_idata, mem_size);

	hipMemcpy(d_idata, data, mem_size, hipMemcpyHostToDevice);

	fwt_1D(&d_idata, level, nx, ny);

	hipMemcpy(data, d_idata, mem_size, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	printf("FWT_GPU: %s\n",hipGetErrorString(hipGetLastError()));

	hipFree(d_idata);
}

extern "C" void iwt_1D_GPU(float *data, const unsigned level, const unsigned nx, const unsigned ny) {
	const int mem_size = nx*ny*sizeof(float);

	float *d_idata;
	hipMalloc(&d_idata, mem_size);

	hipMemcpy(d_idata, data, mem_size, hipMemcpyHostToDevice);

	iwt_1D(&d_idata, level, nx, ny);

	hipMemcpy(data, d_idata, mem_size, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	printf("IWT_GPU: %s\n",hipGetErrorString(hipGetLastError()));

	hipFree(d_idata);
}

// data_dest: set this to the device address where the output data resides.
// If this is not equal to 'data', then 'data' has been freed.
extern "C" void wavelet_cuda_3d_fwd(float *data, const unsigned nx, const unsigned ny, const unsigned nz, const unsigned lvlx, const unsigned lvly, const unsigned lvlz, bool data_is_on_gpu) {
	const int mem_size = nx*ny*nz*sizeof(float);

	float *d_idata, *d_tdata;
	hipMalloc(&d_tdata, mem_size);
        hipMalloc(&d_idata, mem_size);

        if (data_is_on_gpu) {
          hipMemcpy(d_idata, data, mem_size, hipMemcpyDeviceToDevice);
        } else {
          hipMemcpy(d_idata, data, mem_size, hipMemcpyHostToDevice);
        }

	fwt_1D(&d_idata, lvlx, nx, ny*nz);

	transpose(d_tdata, d_idata, nx, ny*nz);

	fwt_1D(&d_tdata, lvly, ny, nz*nx);

	transpose(d_idata, d_tdata, ny, nz*nx);

	fwt_1D(&d_idata, lvlz, nz, nx*ny);

        if (data_is_on_gpu) {
          hipMemcpy(data, d_idata, mem_size, hipMemcpyDeviceToDevice);
        } else {
          hipMemcpy(data, d_idata, mem_size, hipMemcpyDeviceToHost);
        }
        hipFree(d_idata);

	//hipDeviceSynchronize();
	//printf("comp: %s\n",hipGetErrorString(hipGetLastError()));

	hipFree(d_tdata);
}

extern "C" void wavelet_cuda_3d_back(float *data, const unsigned nx, const unsigned ny, const unsigned nz, const unsigned lvlx, const unsigned lvly, const unsigned lvlz) {
	const int mem_size = nx*ny*nz*sizeof(float);

	float *d_idata, *d_tdata;
	hipMalloc(&d_idata, mem_size);
	hipMalloc(&d_tdata, mem_size);

	hipMemcpy(d_idata, data, mem_size, hipMemcpyHostToDevice);

	iwt_1D(&d_idata, lvlz, nz, nx*ny);

	transpose(d_tdata, d_idata, nz*nx, ny);

	iwt_1D(&d_tdata, lvly, ny, nz*nx);

	transpose(d_idata, d_tdata, ny*nz, nx);

	iwt_1D(&d_idata, lvlx, nx, ny*nz);


	hipMemcpy(data, d_idata, mem_size, hipMemcpyDeviceToHost);

	//hipDeviceSynchronize();
	//printf("ucomp: %s\n",hipGetErrorString(hipGetLastError()));

	hipFree(d_idata);
	hipFree(d_tdata);
}
