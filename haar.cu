#include "hip/hip_runtime.h"
#include <cmath>
#include "hip/hip_runtime.h"
#include "cucheck.h"
#include "data_io.h"
#include "nixtimer.h"
#include "dwt_cpu.h"
#include "dwt_gpu.h"
#include "transpose_gpu.h"


bool similar(float a, float b) {
  return fabs(a-b) < .00001;
}


void printHelp() {
  fprintf(stderr, 
          "\n"
          "  haar [-inverse] [-text] [-blocksize]<input datafile> <output datafile> [steps]\n"
          "  Do a Haar discrete wavelet transform.\n"
          "    -inverse : invert the transform\n"
          "    -text : output data in text format rather than binary\n"
          "    -blocksize : specify the thread block size\n"
          "  By default, one transformation step will be done.\n"
          "\n");
  exit(1);
}


int getBestThreadBlockSize(int imageSize) {
  if (imageSize >= 4096) {
    return 1024;
  } else if (imageSize < 512) {
    return 128;
  } else {
    // round imageSize/4 to the nearest power of 2
    return 1 << (int)(log2((double)imageSize) - 2 + .5);
  }
}


int main(int argc, char **argv) {
  if (argc < 3) printHelp();

  bool inverse = false, textOutput = false;
  int argNo = 1, blockSize = -1;

  while (argNo < argc && argv[argNo][0] == '-') {
    if (!strcmp(argv[argNo], "-inverse")) {
      inverse = true;
      argNo++;
    }

    else if (!strcmp(argv[argNo], "-text")) {
      textOutput = true;
      argNo++;
    }

    else if (!strcmp(argv[argNo], "-blocksize")) {
      if (argNo >= argc) printHelp();
      if (1 != sscanf(argv[++argNo], "%d", &blockSize) ||
          blockSize < 1) {
        printf("Invalid block size \"%s\"\n", argv[argNo]);
        return 1;
      }
      argNo++;
    }

    else printHelp();
  }

  // not enough arguments for the input file and output file
  if (argNo+2 > argc) printHelp();

  // read the input file
  const char *inputFilename = argv[argNo++];
  const char *outputFilename = argv[argNo++];
  int stepCount = 1;

  if (argNo < argc) {
    const char *stepsArg = argv[argNo++];
    if (1 != sscanf(stepsArg, "%d", &stepCount)) {
      printf("Invalid step count: \"%s\"\n", stepsArg);
      return 1;
    }
  }
  if (argNo < argc) printHelp();

  float *data_cpu, *data_gpu, elapsed;
  int width, height;
  printf("Reading %s...", inputFilename);
  fflush(stdout);
  if (!readDataFile(inputFilename, &data_cpu, &width, &height)) return 1;
  printf("%d x %d\n", width, height);
  fflush(stdout);

  if (width != height) {
    printf("Error: only square data is currently supported.\n");
    return 1;
  }

  int size = width;
  
  CUCHECK(hipSetDevice(0));

  // Make a copy of the data for the GPU to use.
  // Allocate page-locked virtual memory (that won't be moved from its
  // position in physical memory) so the data can be copied to the GPU
  // via DMA This approximately double the throughput.  Just be sure
  // to free the data with hipHostFree() rather than delete[].
  CUCHECK(hipHostMalloc((void**)&data_gpu, size*size*sizeof(float)));
  memcpy(data_gpu, data_cpu, sizeof(float)*size*size);

  // run the CPU version of the algorithm
  printf("CPU: "); fflush(stdout);
  elapsed = haar_not_lifting_2d(size, data_cpu, inverse, stepCount);
  printf("%.3f ms\n", elapsed);

  // run the GPU version of the algorithm
  if (blockSize == -1) blockSize = getBestThreadBlockSize(size);

  elapsed = haar_not_lifting_2d_cuda(size, data_gpu, inverse, stepCount,
                                     blockSize);

  // Alternative implementation using surfaces.
  // For all inputs I tested, this is slightly slower.
  // elapsed = haar_not_lifting_2d_cuda_surfaces(size, data_gpu, inverse,
  // stepCount, blockSize);

  printf("CUDA: %.6f ms\n", elapsed);

  /*
    // try a variety of thread block sizes
  float *data_gpu_copy = new float[size*size];
  memcpy(data_gpu_copy, data_gpu, sizeof(float)*height*width);
  for (int threadBlockSize = 32; threadBlockSize <= 1024; threadBlockSize*=2) {
    printf("Thread block size: %d\n", threadBlockSize);
    memcpy(data_gpu, data_gpu_copy, sizeof(float)*height*width);
    elapsed = haar_not_lifting_2d_cuda(size, data_gpu, inverse, stepCount,
                                       threadBlockSize);
    printf("CUDA: %.6f ms\n", elapsed);
  }
  delete[] data_gpu_copy
  */

  double totalErr = 0;
  for (int i=0; i < size*size; i++) {
    totalErr += fabs(data_cpu[i] - data_gpu[i]);
  }
  
  double averageErr = totalErr / (size*size);

  if (averageErr < 0.000001) {

    // if the CPU version and the GPU version produced similar results,
    // output the requested file.
    writeDataFile(outputFilename, data_gpu, size, size, !textOutput);
    printf("Wrote %s\n", outputFilename);

  } else {

    // if the results look bad, output two files; one with the CPU results
    // and one with the GPU results.
    printf("Average error = %.7f\n", averageErr);
    
    /*
      printf("CPU:\n");
      printMatrix(width, height, data_cpu);
      
      printf("GPU:\n");
      printMatrix(width, height, data_gpu);
    */
    
    writeDataFile("err_cpu.data", data_cpu, size, size, !textOutput);
    writeDataFile("err_gpu.data", data_gpu, size, size, !textOutput);
    printf("Wrote err_cpu.data and err_gpu.data\n");
  }

  delete[] data_cpu;
  CUCHECK(hipHostFree(data_gpu));

  return 0;
}
