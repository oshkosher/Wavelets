#include "hip/hip_runtime.h"
#include <cmath>
#include "hip/hip_runtime.h"
#include "cucheck.h"
#include "data_io.h"
#include "nixtimer.h"
#include "dwt_cpu.h"
#include "dwt_gpu.h"
#include "transpose_gpu.h"

#define NUM float

bool similar(NUM a, NUM b) {
  return fabs(a-b) < .00001;
}


void printHelp() {
  fprintf(stderr, 
          "\n"
          "  haar [options] <input datafile> <output datafile> [steps]\n"
          "  Do a Haar discrete wavelet transform.\n"
          "  Options:\n"
          "    -inverse : invert the transform\n"
          "    -text : output data in text format rather than binary\n"
          "    -blocksize : specify the thread block size\n"
          "    -gpu <id>|list : specify the GPU to use, or list all\n"
          "  By default, one transformation step will be done.\n"
          "\n");
  exit(1);
}


int getBestThreadBlockSize(int imageSize) {
  if (imageSize >= 4096) {
    return 1024;
  } else if (imageSize < 512) {
    return 128;
  } else {
    // round imageSize/4 to the nearest power of 2
    return 1 << (int)(log2((double)imageSize) - 2 + .5);
  }
}


void listGpus() {
  int gpuCount;
  hipDeviceProp_t prop;
  CUCHECK(hipGetDeviceCount(&gpuCount));

  for (int gpuId=0; gpuId < gpuCount; gpuId++) {
    CUCHECK(hipGetDeviceProperties(&prop, gpuId));
    printf("GPU %d: %s, %.1f MHz, %d MB\n", 
           gpuId, prop.name, prop.clockRate / 1000.0, 
           (int)(prop.totalGlobalMem / (1024*1024)));
  }
}


int main(int argc, char **argv) {
  if (argc < 3) printHelp();

  bool inverse = false, textOutput = false;
  int argNo = 1, blockSize = -1, gpuId = 0;

  int gpuCount;
  CUCHECK(hipGetDeviceCount(&gpuCount));

  while (argNo < argc && argv[argNo][0] == '-') {
    if (!strcmp(argv[argNo], "-inverse")) {
      inverse = true;
      argNo++;
    }

    else if (!strcmp(argv[argNo], "-text")) {
      textOutput = true;
      argNo++;
    }

    else if (!strcmp(argv[argNo], "-blocksize")) {
      if (argNo >= argc) printHelp();
      if (1 != sscanf(argv[++argNo], "%d", &blockSize) ||
          blockSize < 1) {
        printf("Invalid block size \"%s\"\n", argv[argNo]);
        return 1;
      }
      argNo++;
    }

    else if (!strcmp(argv[argNo], "-gpu")) {
      if (argNo >= argc) printHelp();
      argNo++;
      if (!strcmp(argv[argNo], "list")) {
        listGpus();
        return 0;
      } else {
        if (1 != sscanf(argv[argNo], "%d", &gpuId)
            || gpuId < 0
            || gpuId >= gpuCount) {
          printf("Invalid gpu id \"%s\"\n", argv[argNo]);
          return 1;
        }
      }
      argNo++;
    }

    else printHelp();
  }

  // not enough arguments for the input file and output file
  if (argNo+2 > argc) printHelp();

  // read the input file
  const char *inputFilename = argv[argNo++];
  const char *outputFilename = argv[argNo++];
  int stepCount = 1;

  if (argNo < argc) {
    const char *stepsArg = argv[argNo++];
    if (1 != sscanf(stepsArg, "%d", &stepCount)) {
      printf("Invalid step count: \"%s\"\n", stepsArg);
      return 1;
    }
  }
  if (argNo < argc) printHelp();

  NUM *data_cpu, *data_gpu, elapsed;
  int width, height;
  printf("Reading %s...", inputFilename);
  fflush(stdout);
  if (!readDataFile(inputFilename, &data_cpu, &width, &height)) return 1;
  printf("%d x %d\n", width, height);
  fflush(stdout);

  if (width != height) {
    printf("Error: only square data is currently supported.\n");
    return 1;
  }

  int size = width;
  
  CUCHECK(hipSetDevice(gpuId));
  hipDeviceProp_t prop;
  CUCHECK(hipGetDeviceProperties(&prop, gpuId));
  printf("GPU %d: %s\n", gpuId, prop.name);

  // Make a copy of the data for the GPU to use.
  // Allocate page-locked virtual memory (that won't be moved from its
  // position in physical memory) so the data can be copied to the GPU
  // via DMA This approximately double the throughput.  Just be sure
  // to free the data with hipHostFree() rather than delete[].
  CUCHECK(hipHostMalloc((void**)&data_gpu, size*size*sizeof(NUM)));
  memcpy(data_gpu, data_cpu, sizeof(NUM)*size*size);

  // run the CPU version of the algorithm
  printf("CPU: "); fflush(stdout);
  elapsed = haar_2d(data_cpu, size, size, inverse, stepCount);
  printf("%.3f ms\n", elapsed);

  // run the GPU version of the algorithm
  if (blockSize == -1) blockSize = getBestThreadBlockSize(size);

  elapsed = haar_2d_cuda(size, data_gpu, inverse, stepCount,
                                     blockSize, false);

  // Alternative implementation using surfaces.
  // For all inputs I tested, this is slightly slower.
  // elapsed = haar_not_lifting_2d_cuda_surfaces(size, data_gpu, inverse,
  // stepCount, blockSize);

  printf("CUDA: %.6f ms\n", elapsed);

  /*
    // try a variety of thread block sizes
  NUM *data_gpu_copy = new NUM[size*size];
  memcpy(data_gpu_copy, data_gpu, sizeof(NUM)*height*width);
  for (int threadBlockSize = 32; threadBlockSize <= 1024; threadBlockSize*=2) {
    printf("Thread block size: %d\n", threadBlockSize);
    memcpy(data_gpu, data_gpu_copy, sizeof(NUM)*height*width);
    elapsed = haar_not_lifting_2d_cuda(size, data_gpu, inverse, stepCount,
                                       threadBlockSize);
    printf("CUDA: %.6f ms\n", elapsed);
  }
  delete[] data_gpu_copy
  */

  double totalErr = 0;
  for (int i=0; i < size*size; i++) {
    totalErr += fabs(data_cpu[i] - data_gpu[i]);
  }
  
  double averageErr = totalErr / (size*size);

  if (averageErr < 0.000001) {

    // if the CPU version and the GPU version produced similar results,
    // output the requested file.
    writeDataFile(outputFilename, data_gpu, size, size, !textOutput);
    printf("Wrote %s\n", outputFilename);

  } else {

    // if the results look bad, output two files; one with the CPU results
    // and one with the GPU results.
    printf("Average error = %.7f\n", averageErr);
    
    /*
      printf("CPU:\n");
      printMatrix(width, height, data_cpu);
      
      printf("GPU:\n");
      printMatrix(width, height, data_gpu);
    */
    
    writeDataFile("err_cpu.data", data_cpu, size, size, !textOutput);
    writeDataFile("err_gpu.data", data_gpu, size, size, !textOutput);
    printf("Wrote err_cpu.data and err_gpu.data\n");
  }

  delete[] data_cpu;
  CUCHECK(hipHostFree(data_gpu));

  return 0;
}
