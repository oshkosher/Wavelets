#include "hip/hip_runtime.h"
/*
  CUDA implementation of Haar discrete wavelet transform.

  Ed Karrels, ed.karrels@gmail.com, June 2014
*/

#include "dwt_cpu.h"
#include "dwt_gpu.h"
#include "transpose_gpu.h"
#include "cucheck.h"
#include "nixtimer.h"
#include "cuda_timer.h"

#define SQRT2     1.4142135623730950488f
#define INV_SQRT2 0.70710678118654752440f

/*
  To see a previous version of this code that tried out surfaces and
  iterating down columns (rather than rows), see the version as checked
  into Git history as checkin "736c52c":
    git show 736c52c:dwt_gpu.cu | less
*/


/*
  Call structure:

  haar_2d_cuda
    haar_2d_cuda_internal
      haar_2d_kernel
      haar_inv_2d_kernel
      haar_transpose_2d_kernel
      haar_inv_transpose_2d_kernel

*/

template<typename NUM>
float haar_2d_cuda_internal
(int size, NUM *data, bool inverse, int stepCount, int threadBlockSize,
 bool useCombinedTranspose);


/*
  This does a Haar discrete wavelet transform on each row of
  a 2-d array. Each thread block processes one row.
  All data is in global memory.
  Input data is in data[], results will be in result[].
*/
template<typename NUM>
__global__ void haar_2d_kernel
(int arrayWidth, int transformLength, NUM *data, NUM *result) {

  // each thread block processes one row of data
  int y = blockIdx.x;

  // make pointers to my row of data
  NUM *inputRow = data + y * arrayWidth;
  NUM *outputRow = result + y * arrayWidth;

  // Set s to point to my row in the output data
  NUM *s = outputRow;

  int half = transformLength >> 1;
  
  // point d at the second half of the temporary row
  NUM *d = s + half;
  
  for (int i=threadIdx.x; i < half; i += blockDim.x) {
    NUM a = inputRow[2*i], b = inputRow[2*i + 1];
    d[i] = (a - b) * INV_SQRT2;
    s[i] = (a + b) * INV_SQRT2;
  }
}


/* Inverse Haar wavelet transform. */
template<typename NUM>
__global__ void haar_inv_2d_kernel
(int arrayWidth, int transformLength, NUM *data, NUM *result) {

  // each thread block processes one row of data
  int y = blockIdx.x;

  // make pointers to my row of data
  NUM *inputRow = data + y * arrayWidth;
  NUM *outputRow = result + y * arrayWidth;

  // Set s to point to my row in the input data
  NUM *s = inputRow;

  int half = transformLength >> 1;

  // point d at the second half of the temporary row
  NUM *d = s + half;

  for (int i=threadIdx.x; i < half; i += blockDim.x) {
    outputRow[2*i]   = INV_SQRT2 * (s[i] + d[i]);
    outputRow[2*i+1] = INV_SQRT2 * (s[i] - d[i]);
  }
}


/*
  Do one pass of a Haar discrete wavelet transform and transpose the
  matrix at the same time.  This splits the data into tiles, computing
  the transform and writing the results into a transposed matrix.

  In these diagrams, the number indicates the thread that reads
  or writes each element. It depicts each thread block as a 4x4 block
  of 16 threads, but in reality it will be larger--16x16 or 32x32.

  Input matrix (global memory)
  +----------------------------------------
  |  0  0  1  1  2  2  3  3  ...next tile...
  |  4  4  5  5  6  6  7  7
  |  8  8  9  9 10 10 11 11
  | 12 12 13 13 14 14 15 15
  | ...next tile...
   
  Shared memory temporary storage
  The results from each "sum" operation are stored in one 2d array,
  and the results form each "difference" operation are stored in a
  different 2d array.

    Write in this order:
     0  1  2  3
     4  5  6  7
     8  9 10 11
    12 13 14 15

    Read in this order:
    0  4  8 12
    1  5  9 13
    2  6 10 14
    3  7 11 15

  Output matrix (global memory)
  +----------------------------------------
  |  0  1  2  3    ...next tile...
  |  4  5  6  7
  |  8  9 10 11
  | 12 13 14 15
  | ...
  | ...
  | ...lower half....
  |  0  1  2  3
  |  4  5  6  7
  |  8  9 10 11
  | 12 13 14 15
  | ...

Success!
Before:
  Transform time:        40.707 ms (2 calls)
  Transpose time:        57.732 ms (2 calls)
After:
  Transform time:        52.512 ms (2 calls)
  Transpose time:         0.004 ms (2 calls)
 */

template<typename NUM>
__global__ void haar_transpose_2d_kernel
(int arrayWidth, int transformLength, NUM *data, NUM *result,
 int tileSize) {

  // dynamically-sized shared memory
  extern __shared__ int shared[];

  // assign parts of shared memory to my arrays
  NUM *sums, *diffs;
  sums = (NUM*) shared;
  diffs = sums + tileSize * (tileSize+1);

  int inputx = (blockIdx.x*blockDim.x + threadIdx.x) * 2;
  int inputy = blockIdx.y*blockDim.y + threadIdx.y;
  
  // read a tile 2*tileSize wide, tileSize tall, compute
  // the sum and difference coefficients, and store those coefficients
  // transposed in the sums and diffs shared memory arrays.
  int readIdx = inputy * arrayWidth + inputx;

  if (inputx+1 < transformLength && inputy < transformLength) {
    NUM a = data[readIdx], b = data[readIdx+1];
    int shidx = threadIdx.x + threadIdx.y*(tileSize+1);
    sums [shidx] = (a + b) * INV_SQRT2;
    diffs[shidx] = (a - b) * INV_SQRT2;
  }

  __syncthreads();

  // Read the transposed sums and diffs shared memory arrays,
  // and write the data to a tile whose position has been transposed
  int writey = blockIdx.x*blockDim.x + threadIdx.y;
  int writex = blockIdx.y*blockDim.y + threadIdx.x;
  if (writex < transformLength && writey*2 < transformLength) {
    int writeIdx = writey * arrayWidth + writex;
    int shidx = threadIdx.y + threadIdx.x*(tileSize+1);
    result[writeIdx] = sums[shidx];
    writeIdx += arrayWidth*(transformLength>>1);
    result[writeIdx] = diffs[shidx];
  }
}

template<typename NUM>
__global__ void haar_inv_transpose_2d_kernel
(int arrayWidth, int transformLength, NUM *data, NUM *result, int tileSize) {

  // dynamically-sized shared memory
  extern __shared__ int shared[];

  // assign parts of shared memory to my arrays
  NUM *v1, *v2;
  v1 = (NUM*) shared;
  v2 = v1 + tileSize * (tileSize+1);

  int inputx = blockIdx.x*blockDim.x + threadIdx.x;
  int inputy = blockIdx.y*blockDim.y + threadIdx.y;

  // Read the sum and difference coefficients, where the difference coeff
  // is in the second half of the array. Compute the original values v1 and v2,
  // and store them in two shared memory arrays.
  int readIdx1 = inputy * arrayWidth + inputx;
  int readIdx2 = readIdx1 + (transformLength>>1);
  if (inputx < (transformLength>>1) && inputy < transformLength) {
    NUM s = data[readIdx1], d = data[readIdx2];
    int shidx = threadIdx.x * (tileSize+1) + threadIdx.y;
    v1[shidx] = (s + d) * INV_SQRT2;
    v2[shidx] = (s - d) * INV_SQRT2;
  }

  __syncthreads();

  // Read the transposed pair of values v1 and v2 from the transposed
  // shared memory arrays, and write the values to a tile tileSize wide
  // and tileSize*2 tall.
  int writex = blockIdx.y*blockDim.y + threadIdx.x;
  int writey = (blockIdx.x*blockDim.x + threadIdx.y) * 2;
  if (writex < transformLength && writey+1 < transformLength) {
    int writeIdx1 = writey * arrayWidth + writex;
    int writeIdx2 = writeIdx1 + arrayWidth;
    int shidx = threadIdx.y * (tileSize+1) + threadIdx.x;
    result[writeIdx1] = v1[shidx];
    result[writeIdx2] = v2[shidx];
  }
}


float haar_2d_cuda
  (int size, float *data, bool inverse, int stepCount, int threadBlockSize,
   bool useCombinedTranspose) {
  return haar_2d_cuda_internal
    (size, data, inverse, stepCount, threadBlockSize, useCombinedTranspose);
}     


// double support was added in version 1.3
#if !defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 130)
float haar_2d_cuda
  (int size, double *data, bool inverse, int stepCount, int threadBlockSize,
   bool useCombinedTranspose) {
  return haar_2d_cuda_internal
    (size, data, inverse, stepCount, threadBlockSize, useCombinedTranspose);
}
#endif


// haar_transpose_2d_kernel and haar_inv_transpose_2d_kernel use tiles
// to optimize the memory access pattern. After testing tile sizes from
// 8x8 to 32x32 on a few different GPUs, here are the sizes that produced
// the best performance:
//
//   GTX 480: 16     (compute level 2.0)
//   GTX 570: 16     (compute level 2.0)
//   K2000M: 16      (compute level 3.0, laptop)
//   GTX 680: 32     (compute level 3.0)
//   GTX 690: 32     (compute level 3.0)
//   Tesla K20c: 32  (compute level 3.5)
//
int bestTileSize() {
  int gpuId;
  hipDeviceProp_t prop;
  CUCHECK(hipGetDevice(&gpuId));
  CUCHECK(hipGetDeviceProperties(&prop, gpuId));

  // Based on the tests listed above, older (Fermi) and smaller (laptop)
  // GPUs seem to work better with 16x16 tiles, but newer regular GPUs
  // are faster with 32x32 tiles.
  if (prop.major <= 2 || prop.multiProcessorCount <= 2)
    return 16;
  else
    return 32;
}


// Wrapper function that handles the CUDA details.
template<typename NUM>
float haar_2d_cuda_internal
(int size, NUM *data, bool inverse, int stepCount, int threadBlockSize,
 bool useCombinedTranspose) {

  int tileSize = bestTileSize();

  if (useCombinedTranspose) printf("Tile size %dx%d\n", tileSize, tileSize);

  int maxSteps = dwtMaximumSteps(size);
  if (stepCount < 1 || stepCount > maxSteps)
    stepCount = maxSteps;

  // create timers
  CudaTimer overallTimer, copyToTimer, copyFromTimer, 
    transformTimer, transposeTimer;

  // allocate memory for the data and the temp space on the GPU
  NUM *data1_dev, *data2_dev;
  size_t totalBytes = size * size * sizeof(NUM);
  CUCHECK(hipMalloc((void**) &data1_dev, totalBytes));
  CUCHECK(hipMalloc((void**) &data2_dev, totalBytes));

  // Create a stream to enable asynchronous operation, to minimize
  // time between kernel calls.
  hipStream_t stream = 0;
  CUCHECK(hipStreamCreate(&stream));

  // start the timer
  double startTimeCPU = NixTimer::time();
  overallTimer.start(stream);

  // copy the data to the GPU
  copyToTimer.start(stream);
  CUCHECK(hipMemcpyAsync(data1_dev, data, totalBytes, hipMemcpyHostToDevice,
                          stream));
  copyToTimer.end(stream);

  size_t sharedMemSize = tileSize * (tileSize+1)
    * 2 * sizeof(float);
  
  int transformLength;

  if (inverse) {

    // inverse
    transformLength = size >> (stepCount - 1);
    for (int i=0; i < stepCount; i++) {

      dim3 gridDim((transformLength - 1) / (tileSize*2) + 1,
                   (transformLength - 1) / (tileSize) + 1);
      dim3 blockDim(tileSize, tileSize);

      if (useCombinedTranspose) {

        // transform columns and transpose
        transformTimer.start(stream);
        haar_inv_transpose_2d_kernel
          <<<gridDim, blockDim, sharedMemSize, stream>>>
          (size, transformLength, data1_dev, data2_dev, tileSize);
        transformTimer.end(stream);
    
        // transform rows and transpose
        transformTimer.start(stream);
        haar_inv_transpose_2d_kernel
          <<<gridDim, blockDim, sharedMemSize, stream>>>
          (size, transformLength, data2_dev, data1_dev, tileSize);
        transformTimer.end(stream);

      } else {

        // transform columns
        transformTimer.start(stream);
        haar_inv_2d_kernel
          <<<transformLength, threadBlockSize, 0, stream>>>
          (size, transformLength, data1_dev, data2_dev);
        transformTimer.end(stream);

        // transpose the matrix into temp_dev
        transposeTimer.start(stream);
        gpuTranspose(size, transformLength, data2_dev, data1_dev, stream);
        transposeTimer.end(stream);
    
        // transform rows
        transformTimer.start(stream);
        haar_inv_2d_kernel
          <<<transformLength, threadBlockSize, 0, stream>>>
          (size, transformLength, data1_dev, data2_dev);
        transformTimer.end(stream);

        // transpose the matrix into data_dev
        transposeTimer.start(stream);
        gpuTranspose(size, transformLength, data2_dev, data1_dev, stream);
        transposeTimer.end(stream);

        // results are in data1_dev

      }

      transformLength <<= 1;
    }

  } else {

    // forward
    transformLength = size;
    
    for (int i=0; i < stepCount; i++) {

      dim3 gridDim((transformLength - 1) / (tileSize*2) + 1,
                   (transformLength - 1) / (tileSize) + 1);
      dim3 blockDim(tileSize, tileSize);
    
      if (useCombinedTranspose) {

        // do the wavelet transform on rows
        transformTimer.start(stream);
        haar_transpose_2d_kernel
          <<<gridDim, blockDim, sharedMemSize, stream>>>
          (size, transformLength, data1_dev, data2_dev, tileSize);
        transformTimer.end(stream);

        // do the wavelet transform on columns
        transformTimer.start(stream);
        haar_transpose_2d_kernel
          <<<gridDim, blockDim, sharedMemSize, stream>>>
          (size, transformLength, data2_dev, data1_dev, tileSize);
        transformTimer.end(stream);

      } else {

        // do the wavelet transform on rows
        transformTimer.start(stream);
        haar_2d_kernel
          <<<transformLength, threadBlockSize, 0, stream>>>
          (size, transformLength, data1_dev, data2_dev);
        transformTimer.end(stream);

        // transpose the matrix into temp_dev
        transposeTimer.start(stream);
        gpuTranspose(size, transformLength, data2_dev, data1_dev, stream);
        transposeTimer.end(stream);
    
        // do the wavelet transform on columns
        transformTimer.start(stream);
        haar_2d_kernel
          <<<transformLength, threadBlockSize, 0, stream>>>
          (size, transformLength, data1_dev, data2_dev);
        transformTimer.end(stream);
    
        // transpose the matrix back into data_dev
        transposeTimer.start(stream);
        gpuTranspose(size, transformLength, data2_dev, data1_dev, stream);
        transposeTimer.end(stream);

      }

      transformLength >>= 1;
    }

  }

  // copy the data back from the GPU
  copyFromTimer.start(stream);
  CUCHECK(hipMemcpyAsync(data, data1_dev, totalBytes, hipMemcpyDeviceToHost,
                          stream));
  copyFromTimer.end(stream);


  // Since all the GPU tasks were started asynchronously, control should
  // flow to this point very quickly. The hipEventSynchronize() call will
  // wait until the GPU is finished.
  double endTimeCPU = NixTimer::time();
  printf("Time elapsed creating GPU tasks: %.3f ms\n",
         1000*(endTimeCPU - startTimeCPU));
  fflush(stdout);

  // stop the timer
  overallTimer.end(stream);
  CUCHECK(hipEventSynchronize(overallTimer.getLastEvent()));
  hipStreamDestroy(stream);

  // check for errors
  CUCHECK(hipGetLastError());

  printf("Times:\n");
  printf("  Copy data to GPU:   %9.3f ms\n", copyToTimer.time());
  printf("  Transform time:     %9.3f ms (%d calls)\n",
         transformTimer.time(), transformTimer.count());
  if (transposeTimer.count() > 0) {
    printf("  Transpose time:     %9.3f ms (%d calls)\n", 
           transposeTimer.time(), transposeTimer.count());
  }
  printf("  Copy data from GPU: %9.3f ms\n", copyFromTimer.time());

  // deallocate GPU memory
  CUCHECK(hipFree(data1_dev));
  CUCHECK(hipFree(data2_dev));

  return overallTimer.time();
}
