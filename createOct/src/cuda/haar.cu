#include "hip/hip_runtime.h"
#include <cmath>
#include "hip/hip_runtime.h"
#include "cucheck.h"
#include "dwt_gpu.h"

void printHelp() {
  fprintf(stderr, 
          "\n"
          "  haar [options] <input datafile> <output datafile> [steps]\n"
          "  Do a Haar discrete wavelet transform.\n"
          "  Options:\n"
          "    -inverse : invert the transform\n"
          "    -text : output data in text format rather than binary\n"
          "    -blocksize : specify the thread block size\n"
          "    -gpu <id>|list : specify the GPU to use, or list all\n"
          "  By default, one transformation step will be done.\n"
          "\n");
  exit(1);
}

int getBestThreadBlockSize(int imageSize) {
  if (imageSize >= 4096) {
    return 1024;
  } else if (imageSize < 512) {
    return 128;
  } else {
    // round imageSize/4 to the nearest power of 2
    return 1 << (int)(log2((double)imageSize) - 2 + .5);
  }
}

int haar(float *output, float *input, int width, int steps, bool inverse, int blockSize) {

  float *plmemory;
  float elapsed;
  
  hipDeviceProp_t prop;
  CUCHECK(hipGetDeviceProperties(&prop, 0));
  printf("GPU %d: %s\n", 0, prop.name);

  // Make a copy of the data for the GPU to use.
  // Allocate page-locked virtual memory (that won't be moved from its
  // position in physical memory) so the data can be copied to the GPU
  // via DMA This approximately double the throughput.  Just be sure
  // to free the data with hipHostFree() rather than delete[].
  CUCHECK(hipHostMalloc((void**)&plmemory, width*width*sizeof(float)));
  memcpy(plmemory, input, sizeof(float)*width*width);

  // run the GPU version of the algorithm
  if (blockSize == -1) blockSize = getBestThreadBlockSize(width);

  elapsed = haar_not_lifting_2d_cuda(width, plmemory, inverse, steps,
                                     blockSize, true);

  memcpy(output, plmemory, sizeof(float)*width*width);

  printf("CUDA: %.6f ms\n", elapsed);

  CUCHECK(hipHostFree(plmemory));

  return 0;
}

int haar(double *output, double *input, int width, int steps, bool inverse, int blockSize) {

  double *plmemory;
  float elapsed;
  
  hipDeviceProp_t prop;
  CUCHECK(hipGetDeviceProperties(&prop, 0));
  printf("GPU %d: %s\n", 0, prop.name);

  // Make a copy of the data for the GPU to use.
  // Allocate page-locked virtual memory (that won't be moved from its
  // position in physical memory) so the data can be copied to the GPU
  // via DMA This approximately double the throughput.  Just be sure
  // to free the data with hipHostFree() rather than delete[].
  CUCHECK(hipHostMalloc((void**)&plmemory, width*width*sizeof(double)));
  memcpy(plmemory, input, sizeof(double)*width*width);

  // run the GPU version of the algorithm
  if (blockSize == -1) blockSize = getBestThreadBlockSize(width);

  elapsed = haar_not_lifting_2d_cuda(width, plmemory, inverse, steps,
                                     blockSize, true);

  memcpy(output, plmemory, sizeof(double)*width*width);

  printf("CUDA: %.6f ms\n", elapsed);

  CUCHECK(hipHostFree(plmemory));

  return 0;
}

