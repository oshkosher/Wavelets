#include "hip/hip_runtime.h"
#include "transpose_gpu.h"

#define TX_BLOCK_SIZE 16

/* Given the width and height of the input matrix, transpose it
   into the given output matrix.

   blockSize is the size of the square thread block that will process
   each tile.
*/
void gpuTranspose(int width, int height, float *matrix_d, float *matrixTx_d,
                  hipStream_t stream) {
                  

  dim3 gridSize, blockSize(TX_BLOCK_SIZE, TX_BLOCK_SIZE);
  gridSize.x = ceil(width-1) / TX_BLOCK_SIZE + 1;
  gridSize.y = ceil(height-1) / TX_BLOCK_SIZE + 1;

  gpuTransposeKernel<<<gridSize, blockSize, 0, stream>>>
    (width, height, matrix_d, matrixTx_d);
}


/* gpuTransposeTiledKernel
   Copy one 16x16 tile (one element per thread) from the first matrix
   into shared memory, then from shared memory to the second matrix.
   Be sure to structure the global memory accesses so that consecutive
   threads access consecutive memory.
*/
__global__ void gpuTransposeKernel(int width, int height, float *matrix,
                                   float *matrixTx) {

  __shared__ float cache[TX_BLOCK_SIZE+1][TX_BLOCK_SIZE+1];
  int tileTop = blockIdx.y * blockDim.y;
  int tileLeft = blockIdx.x * blockDim.x;

  // Since the matrix size is a multiple of TILE_SIZE, as long as the 
  // upper left corner of the tile is in the matrix, the entire tile will be.
  // if (tileTop >= height || tileLeft >= width) return;

  int row = tileTop + threadIdx.y;
  int col = tileLeft + threadIdx.x;
  
  if (row < width && col < height)
    cache[threadIdx.y][threadIdx.x] = matrix[row * width + col];

  // Sync is necessary because the thread that wrote to a shared memory
  // entry won't always be the one that reads from it.
  __syncthreads();

  row = tileLeft + threadIdx.y;
  col = tileTop + threadIdx.x;
  if (row < width && col < height)
    matrixTx[row * height + col] = cache[threadIdx.x][threadIdx.y];
}
