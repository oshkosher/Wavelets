#include "hip/hip_runtime.h"

static float *h_points;
static float *d_points;
static unsigned int *d_groups;
static unsigned int d_psize;
static float d_pmax;
static float d_pmin;

__global__ void groupKernel(float *points, unsigned int psize, float *codebook, unsigned int csize, unsigned int *groups) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < psize) {
		float d = abs(points[idx] - codebook[0]);
		float min = d;
		unsigned int g = 0;
		for(int i = 1; i < csize; i++) {
			d = abs(points[idx] - codebook[i]);
			g += (d < min) * (i - g);
		}
		groups[idx] = g;
	}
}

__global__ void tableKernel(float *points, unsigned int psize, float *codebook, unsigned int csize, unsigned int *groups) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < psize) {
		unsigned int g = csize; // out of range
		for(int i = 1; i < csize; i++) {
			g = (points[idx] > codebook[i]) * i;
		}
		groups[idx] = g;
	}
}

void setupLloyds(float *points, unsigned int psize, float pmax, float pmin) {
	d_psize = psize;
	d_pmax = pmax;
	d_pmin = pmin;

	h_points = points;
  hipMalloc((void**)&d_points, sizeof(float)*psize);
  hipMalloc((void**)&d_groups, sizeof(unsigned int)*psize);

	hipMemcpy(d_points, points, sizeof(float) * psize, hipMemcpyHostToDevice);
}

void lloyd(float *codebook, unsigned int csize, float stop_criteria, float *table, float &dist, float &reldist) {

	float *d_codebook;
	dist = 0;
	reldist = 0;

	unsigned int *groups = (unsigned int*)malloc(sizeof(unsigned int)*d_psize);

	// Calculate initial table
	//can be done in gpu if csize is big enough
	for(int i = 0; i < csize-1; i++) {
		table[i] = (codebook[i] + codebook[i+1]) / 2;
	}

  hipMalloc((void**)&d_codebook, sizeof(float)*csize);
	hipMemcpy(d_codebook, codebook, sizeof(float)*csize, hipMemcpyHostToDevice);

	// Assign each point its codebook group
	unsigned int threads = 256;
	unsigned int blocks = (d_psize - 1) / threads + 1;
	groupKernel<<<blocks, threads>>>(d_points, d_psize, d_codebook, csize, d_groups);

	hipMemcpy(groups, d_groups, sizeof(float) * d_psize, hipMemcpyDeviceToHost);
	
	unsigned int *incode = (unsigned int*)calloc(csize, sizeof(unsigned int));
	float *meancode = (float*)malloc(sizeof(float) * csize);

	// Calculate the mean of each codebook group and distortion
	for(int i = 0; i < d_psize; i++) {
		incode[groups[i]]++;
		meancode[groups[i]] += h_points[i];
		dist += codebook[groups[i]] - h_points[i];
	}
	dist /= d_psize;

	reldist = abs(dist);

	while(reldist > stop_criteria) {

		// Update codebook
		//can be done in gpu if csize is big enough
		for(int i = 0; i < csize; i++) {
			if(incode[i] != 0) {
				codebook[i] = meancode[i] / incode[i];
				incode[i] = 0;
			} else if (i == 0) {
				codebook[i] = (table[0] + d_pmin) / 2;
			} else if (i == csize-1) {
				codebook[i] = (table[i-1] + d_pmax) / 2;
			} else {
				codebook[i] = (table[i-1] + table[i]) / 2;
			}
			meancode[i] = 0;
		}

		hipMemcpy(d_codebook, codebook, sizeof(float)*csize, hipMemcpyHostToDevice);

		// Calculate mean of points between codebooks for table update
		tableKernel<<<blocks, threads>>>(d_points, d_psize, d_codebook, csize, d_groups);

		hipMemcpy(groups, d_groups, sizeof(float) * d_psize, hipMemcpyDeviceToHost);

		for(int i = 0; i < d_psize; i++) {
			incode[groups[i]]++;
			meancode[groups[i]] += h_points[i];
		}

		// Update table
		//can be done in gpu if csize is big enough
		for(int i = 0; i < csize-1; i++) {
			if(incode[i] != 0) {
				table[i] = meancode[i] / incode[i];
				incode[i] = 0;
			} else {
				table[i] = (codebook[i] + codebook[i+1]) / 2;
			}
			meancode[i] = 0;
		}

		// Assign each point its codebook group
		groupKernel<<<blocks, threads>>>(d_points, d_psize, d_codebook, csize, d_groups);

		hipMemcpy(groups, d_groups, sizeof(float) * d_psize, hipMemcpyDeviceToHost);

		// Calculate the mean of each codebook group and distortion
		for(int i = 0; i < d_psize; i++) {
			incode[groups[i]]++;
			meancode[groups[i]] += h_points[i];
			dist += codebook[groups[i]] - h_points[i];
		}
		dist /= d_psize;
	
		reldist = abs(reldist - dist);
	}  // END WHILE

	hipFree(d_codebook);
	free(table);
	free(incode);
	free(meancode);
}


void finalize() {
	hipFree(d_points);
	hipFree(d_groups);
}
